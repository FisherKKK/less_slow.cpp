/**
 *  @brief  Low-level CUDA kernels for building a performance-first mindset.
 *  @file   less_slow.cuh
 *  @author Ash Vardanian
 *
 *  The contents of this file complement the contents of the `less_slow.cpp`
 *  file with GPGPU kernels showcasing:
 *
 *  - How to use Tensor Cores for matrix multiplications?
 *    What's the difference between `mma` and `wgmma` on Hopper?
 *  - TODO: How to coordinate CUDA cores within a single block or warp?
 *    A.k.a. how to use shared memory, warp shuffle intrinsics, and reductions?
 *  - TODO: What are CUDA math intrinsics and how much faster are they?
 *    A.k.a. when to use `__sinf` over `sinf` or `__fdividef` over `a / b`?
 *  - TODO: What's the Physical Page Caching behavior on GPUs?
 *  - TODO: How to schedule advanced computational graphs on GPUs?
 *    A.k.a. CUDA streams vs Graph Node API vs Cooperative Groups?
 *
 *  To compile this file, dump the SASS code, and check for Tensor Cores usage
 *  on Volta SM70 GPUs, use the following commands:
 *
 *  $ nvcc -arch=sm_90 -Xptxas -v -lineinfo -ptx -o less_slow_from_cu.ptx less_slow.cu
 *  $ nvcc -arch=sm_90 -Xptxas -v -lineinfo -cubin -o less_slow_from_cu.cubin less_slow.cu
 *  $ cuobjdump -sass less_slow_from_cu.cubin | grep -i mma
 *
 *  Keep in mind the following TC generations:
 *
 *  - Volta SM70: 1st generation of TCs, server V100 cards.
 *  - Turing SM75: 2nd generation of TCs, consumer RTX 30 cards.
 *  - Ampere SM80: 3rd generation of TCs, server A100 cards.
 *  - Ada Lovelace SM89: 4th generation of TCs, consumer RTX 40 cards.
 *  - Hopper SM90: 5th generation of TCs, server H100 cards.
 *
 *  Looking at server-side V100, A100, and H100 GPUs, most features are
 *  identical, except for @b shared-memory size and TCs:
 *
 *    Feature                              | V100     | A100     | H100
 *    -------------------------------------|----------|----------|----------
 *    Compute Capability                   | 7.0      | 8.0      | 9.0
 *    PTX Version                          | 6+       | 7+       | 8+
 *    CUDA Releases                        | 9-10     | 11+      | 12+
 *    -------------------------------------|----------|----------|----------
 *    Threads / Warp                       | 32       | 32       | 32
 *    Max Warps / SM                       | 64       | 64       | 64
 *    Max Threads / SM                     | 2048     | 2048     | 2048
 *    Max Thread Blocks (CTAs) / SM        | 32       | 32       | 32
 *    Max Thread Blocks / Thread Block Cl. | NA       | NA       | 16
 *    Max 32-bit Registers / SM            | 65536    | 65536    | 65536
 *    Max Registers / Thread Block (CTA)   | 65536    | 65536    | 65536
 *    Max Registers / Thread               | 255      | 255      | 255
 *    Max Thread Block Size (# of threads) | 1024     | 1024     | 1024
 *    -------------------------------------|----------|----------|----------
 *    Ratio of SM Registers to FP32 Cores  | 1024     | 1024     | 512
 *    Shared Memory Size / SM              | ≤ 96 KB  | ≤ 164 KB | ≤ 228 KB
 *    Tensor Core Generation               | 1st      | 3rd      | 5th
 *
 */

#include <hip/hip_runtime.h>
#include <cstdint> // `std::uint8_t`
#if (__CUDA_ARCH__ >= 700)
#include <cuda_fp16.h> // `half` type
#endif
#if (__CUDA_ARCH__ >= 750)
#include <cuda_bf16.h> // `__nv_bfloat16` type
#endif

template <typename scalar_type_, std::size_t side_>
struct small_square_matrix {
    scalar_type_ scalars[side_][side_];
};

/**
 *  @brief  A CUDA kernel that computes the product of two small square matrices.
 *          Doesn't use any block/warp-level communication and optimizations.
 */
template <typename scalar_type_, std::size_t side_>
small_square_matrix<scalar_type_, side_> small_matmul_kernel_cuda( //
    small_square_matrix<scalar_type_, side_> const &a,             //
    small_square_matrix<scalar_type_, side_> const &b) {

    small_square_matrix<scalar_type_, side_> c;
    for (std::size_t i = 0; i != side_; ++i)
        for (std::size_t j = 0; j != side_; ++j)
            for (std::size_t k = 0; k != side_; ++k) c.scalars[i][j] += a.scalars[i][k] * b.scalars[k][j];
    return c;
}

/**
 *  Starting with Nvidia Volta GPUs, specialized "Tensor Cores" @b (TC) are
 *  added for faster matrix multiplications. These Tensor Cores are much faster
 *  than native CUDA implementation of dot-product operations and provide
 *  special intrinsics for programmers to use.
 *
 *  Unlike typical CPU-side intrinsics, in CUDA, C++ templates are used.
 *  There is not a single Tensor Core generation that natively performs
 *  @b 16x16x16 FP16 matrix multiplication into FP32 accumulators.
 *  But we can use @b `wmma` 2D tiles of that size, that will be unpacked
 *  into the right combination of instructions at compile time.
 *
 *  Theoretically, this implies that we could have used 256x256x256 matrices,
 *  or some other size that optimally fits into the GPU's caches, shared along
 *  the cores in the same warp, but @b NO! Most sizes won't compile.
 *
 *  Moreover, splitting into hardware-specific tile sizes isn't done at the PTX
 *  level! It's done at the SASS level, so the PTX output for this kernel will
 *  still contain lines like:
 *
 *  ! wmma.mma.sync.aligned.row.col.m16n16k16.f32.f32 {}, {}, {}, {};
 *
 *  That will be lowered to the right SASS instructions by the PTXAS assembler,
 *  and on Volta SM70 GPUs, will use the only supported size of 8x8x4:
 *
 *  ! HMMA.884.F32.F32.STEP2 R8, R2.reuse.ROW, R2.reuse.COL, R8
 *
 *  Unpacking it:
 *  - HMMA stands for Half-precision Matrix Multiply & Accumulate.
 *  - 884 stands for the 8x8x4 shape of the matrix multiplication.
 *  - F32.F32 defines the multiplication and accumulation precision.
 *  - STEPx denotes the stage of the computation for a specific tile, where
 *    each HMMA instruction contributes to completing a part of the final
 *    result. In our case we will get 4 STEPs, repeated 4 times, for a
 *    total of 16x HMMA instructions per WMMA intrinsic.
 *
 *  For optimal usage of Tensor Cores:
 *  - Ensure your matrix dimensions are multiples of the tile size (8x8x4 on Volta).
 *  - Use shared memory efficiently to reduce global memory accesses.
 *  - Properly align input and output matrices in memory (128-byte alignment).
 *
 *  @see Supported numeric types until Ampere SM80:
 *       https://docs.nvidia.com/cuda/ampere-tuning-guide/index.html#improved-tensor-core-operations
 *  @see "Benchmarking and Dissecting the Nvidia Hopper GPU Architecture" paper
 *       from HKSTU: https://arxiv.org/pdf/2402.13499v1
 *
 */
#include <mma.h> // `mma::` intrinsics

/**
 *  @brief  A CUDA kernel that @b repeatedly computes the product of two small
 *          matrices of size MxN and NxK using Tensor Cores.
 */
template <typename input_type_, typename output_type_, int m_, int n_, int k_, int repetitions_>
__device__ inline void tops_tc_cuda_kernel() {
    using namespace nvcuda;
    wmma::fragment<wmma::matrix_a, m_, n_, k_, input_type_, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, m_, n_, k_, input_type_, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, m_, n_, k_, output_type_> c_frag;

    // To initialize, we can call:
    //
    //      wmma::fill_fragment(a_frag, 1);
    //      wmma::fill_fragment(b_frag, 1);
    //      wmma::fill_fragment(c_frag, 0);
    //
    // To better saturate the ALU, we could unroll a few iterations:
    for (int i = 0; i != repetitions_; ++i) wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

    // Impossible condition to prevent optimization
    if (threadIdx.x == 2147483647) wmma::store_matrix_sync(nullptr, c_frag, 16, wmma::mem_row_major);
}

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750) //? Binary Matrices require SM75 or higher

/**
 *  To process binary matrices we can't rely on addition and multiplication.
 *  A different set of mathematical operations is required, such as @b XOR or
 *  @b AND as multiplication and @b POPCOUNT as accumulation. The names of
 *  those operations are passed as extra arguments to the @b `bmma_sync`.
 *
 *  @see Docs: https://docs.nvidia.com/cuda/cuda-c-programming-guide/#sub-byte-operations
 */
template <typename input_type_, typename output_type_, int m_, int n_, int k_, int repetitions_>
__device__ inline void binary_tops_tc_cuda_kernel( //
    nvcuda::wmma::experimental::bmmaBitOp bit_op, nvcuda::wmma::experimental::bmmaAccumulateOp acc_op) {
    using namespace nvcuda;
    wmma::fragment<wmma::matrix_a, m_, n_, k_, input_type_, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, m_, n_, k_, input_type_, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, m_, n_, k_, output_type_> c_frag;
    for (int i = 0; i != repetitions_; ++i) wmma::bmma_sync(c_frag, a_frag, b_frag, c_frag, bit_op, acc_op);
    if (threadIdx.x == 2147483647) wmma::store_matrix_sync(nullptr, c_frag, 16, wmma::mem_row_major);
}

#endif

#pragma region Volta

__global__ void tops_f16f16_sm70tc_16x16x16_1024unroll_cuda_kernel() {
    //? On Volta: 8x8x4.
    //? On Turing: 8x8x4 / 16x8x8 / 16x8x16.
    //? On Ampere: 16x8x8 / 16x8x16.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 700)
    tops_tc_cuda_kernel<half, half, 16, 16, 16, 1024>();
#endif
}
__global__ void tops_f16f32_sm70tc_16x16x16_1024unroll_cuda_kernel() {
    //? On Volta: 8x8x4.
    //? On Turing: 8x8x4 / 16x8x8 / 16x8x16.
    //? On Ampere: 16x8x8 / 16x8x16.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 700)
    tops_tc_cuda_kernel<half, float, 16, 16, 16, 1024>();
#endif
}

#pragma endregion

#pragma region Turing

__global__ void tops_u8i32_sm75tc_16x16x16_1024unroll_cuda_kernel() {
    //? On Turing: 8x8x16.
    //? On Ampere: 8x8x16 / 16x8x16 / 16x8x32.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750)
    tops_tc_cuda_kernel<std::uint8_t, int32_t, 16, 16, 16, 1024>();
#endif
}
__global__ void tops_u4i32_sm75tc_8x8x32_1024unroll_cuda_kernel() {
    //! The 16x16x16 won't compile, 8x8x32 will.
    //? On Turing: 8x8x32.
    //? On Ampere: 8x8x32 / 16x8x32 / 16x8x64.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750)
    tops_tc_cuda_kernel<nvcuda::wmma::experimental::precision::u4, int32_t, 8, 8, 32, 1024>();
#endif
}
__global__ void tops_b1i32xor_sm75tc_8x8x128_1024unroll_cuda_kernel() {
    //! The 16x16x16 won't compile, 8x8x128 will.
    //? On Turing: 8x8x128.
    //? On Ampere: 8x8x128 / 16x8x128 / 16x8x256.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750)
    binary_tops_tc_cuda_kernel<nvcuda::wmma::experimental::precision::b1, int32_t, 8, 8, 128, 1024>(
        nvcuda::wmma::experimental::bmmaBitOp::bmmaBitOpXOR,
        nvcuda::wmma::experimental::bmmaAccumulateOp::bmmaAccumulateOpPOPC);
#endif
}

#pragma endregion

#pragma region Ampere

__global__ void tops_bf16f32_sm80tc_16x16x16_1024unroll_cuda_kernel() {
    //? On Ampere: 16x8x8 / 16x8x16.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800)
    tops_tc_cuda_kernel<__nv_bfloat16, float, 16, 16, 16, 1024>();
#endif
}
__global__ void tops_tf32f32_sm80tc_16x16x8_1024unroll_cuda_kernel() {
    //! The 16x16x16 won't compile, 16x16x8 will.
    //? On Ampere: 16x8x4.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800)
    tops_tc_cuda_kernel<nvcuda::wmma::precision::tf32, float, 16, 16, 8, 1024>();
#endif
}
__global__ void tops_f64f64_sm80tc_8x8x4_1024unroll_cuda_kernel() {
    //! The 16x16x16 won't compile, 8x8x4 will.
    //? On Ampere: 8x8x4.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800)
    tops_tc_cuda_kernel<double, double, 8, 8, 4, 1024>();
#endif
}

__global__ void tops_b1i32and_sm80tc_8x8x128_1024unroll_cuda_kernel() {
    //! The 16x16x16 won't compile, 8x8x128 will.
    //? On Ampere: 8x8x128 / 16x8x128 / 16x8x256.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800)
    binary_tops_tc_cuda_kernel<nvcuda::wmma::experimental::precision::b1, int32_t, 8, 8, 128, 1024>(
        nvcuda::wmma::experimental::bmmaBitOp::bmmaBitOpAND,
        nvcuda::wmma::experimental::bmmaAccumulateOp::bmmaAccumulateOpPOPC);
#endif
}

#pragma endregion

/**
 *  MMA is not the only family of tensor core instructions:
 *
 *  - MMA for dense-dense synchronous matrix multiplication.
 *  - Sparse MMA for synchronous sparse-dense matrix multiplication with
 *    a known @b structured sparsity pattern. Those are handy when you have
 *    a portion X of Y consecutive cells equal to zero. X and Y are generally
 *    set to 2 and 4, respectively, for a "2:4" pattern.
 *  - @b WGMMA or Warp-Group MMA operates on 4 contiguous warps, forming 128
 *    contiguous threads, generalizing the original MMA in 2 ways:
 *
 *    1. They can be asynchronous, for more flexible scheduling.
 *    2. They can avoid accumulation, a.k.a $C = A * B$, not $C += A * B$.
 *
 *  The later are vastly more complex. Just compare our old MMA signature:
 *  ! {wmma.mma.sync.aligned}.{row.col}.{m16n16k16}.{f32.f32} { ........ }
 *  ? {        header       }.{ layout}.{  shape  }.{ types } { operands }
 *
 *  To the new WGMMA signature:
 *  ! {wgmma.mm_async.sync.aligned}.{m64n64k16}.{f32.f16.f16} { ........ },{ .... }
 *  ? {     much longer header    }.{  shape  }.{   types   } { operands },{ args }
 *
 *  @see "Fast Matrix-Multiplication with WGMMA on NVIDIA Hopper GPUs" by Colfax:
 *       https://research.colfax-intl.com/cutlass-tutorial-wgmma-hopper/
 *  @see "Outperforming cuBLAS on H100: a Worklog" by Pranjal Shankhdhar:
 *       https://cudaforfun.substack.com/p/outperforming-cublas-on-h100-a-worklog
 */